#include "hip/hip_runtime.h"
#include <cstdio>
#include <ctime>
#include <vector>
#include <algorithm>
#ifdef USE_PTHREADS
#include <pthread.h>
#else
#include <omp.h>
#endif
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
// SRAND48 and DRAND48 don't exist on windows, but these are the equivalent
// functions
void srand48(long seed) { srand((unsigned int)seed); }
double drand48() { return double(rand()) / RAND_MAX; }
#endif

const char *sSDKname = "UnifiedMemoryStreams";

template <typename T>
struct Task {
    unsigned int size, id;
    T *data;
    T *result;
    T *vector;

    Task() : size(0), id(0), data(NULL), result(NULL), vector(NULL) {};
    Task(unsigned int s) : size(s), id(0), data(NULL), result(NULL) {
        // allocate unified memory -- the operation performed in this example will be a DGEMV
        checkCudaErrors(hipMallocManaged(&data, sizeof(T) * size * size));
        checkCudaErrors(hipMallocManaged(&result, sizeof(T) * size));
        checkCudaErrors(hipMallocManaged(&vector, sizeof(T) * size));
        checkCudaErrors(hipDeviceSynchronize());
    }

    ~Task() {
        // ensure all memory is dellocated
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipFree(data));
        checkCudaErrors(hipFree(result));
        checkCudaErrors(hipFree(vector));
    }

    void allocate(const unsigned int s, const unsigned int unique_id) {
        // allocated unified memory outside of constructor
        id = unique_id;
        size = s;
        checkCudaErrors(hipMallocManaged(&data, sizeof(T) * size * size));
        checkCudaErrors(hipMallocManaged(&result, sizeof(T) * size));
        checkCudaErrors(hipMallocManaged(&vector, sizeof(T) * size));
        checkCudaErrors(hipDeviceSynchronize());

        for (unsigned int i = 0; i < size * size; ++i) {
            data[i] = drand48();
        }

        for (unsigned int i = 0; i < size; ++i) {
            result[i] = 0;
            vector[i] = drand48();
        }
    }
};

#ifdef USE_PTHREADS
struct threadData_t {
    int tid;
    Task<double> *TaskListPtr;
    hipStream_t *streams;
    hipblasHandle_t *handles;
    int taskSize;
}

typedef struct threadData_t threadData;
#endif


template <typename T>
void gemv (int m, int n, T alpha, T *A, T *x, T beta, T *result) {
    // simple host dgemv: assume data is in row-major format and square
    // 复杂度是O(n²)
    for (int i = 0; i < n; ++i) {
        result[i] *= beta;
        for (int j = 0; j < n; ++j) {
            result[i] += A[i * n + j] * x[j];
        }
    }
}

// execute a single task on a either host or device depending on size
#ifdef USE_PTHREADS
void *execute(void *inpArgs) {
    threadData *dataPtr = (threadData *)inpArgs;
    hipStream_t *stream = dataPtr->streams;
    hipblasHandle_t *handle = dataPtr->handles;
    int tid = dataPtr->tid;

    for (int i = 0; i < dataPtr->taskSize; ++i) {
        Task<double> &t = dataPtr->TaskListPtr[i];
        if (t.size < 100) {
            printf("Task %d, thread %d executing on host %d. \n", t.id. tid, t.size);
            // 将指定的内存区域与给定的 CUDA 流绑定.这种绑定可以优化数据在 CPU 和 GPU 之间的共享
            // stream[0] 是一个共享的 CUDA 流.
            checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
            checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
            checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
            // necessary to ensure Async hipStreamAttachMemAsync calls have finished
            checkCudaErrors(hipStreamSynchronize(stream[0]));

            gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
        }else {
            // perform on device
            printf("Task [%d], thread [%d] executing on device (%d)\n", t.id, tid, t.size);
            double one = 1.0;
            double zero = 0.0;
            // stream[tid + 1] 是分配给每个线程的独立流, 独立流允许每个线程在 GPU 上并行执行任务,从而实现任务的并发处理.
            checkCudaErrors(hipblasSetStream(handle[tid + 1], stream[tid + 1]));
            checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.data, 0, hipMemAttachSingle));
            checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.vector, 0, hipMemAttachSingle));
            checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.result, 0, hipMemAttachSingle));
            // call the device operation
            checkCudaErrors(hipblasDgemv(handle[tid + 1], HIPBLAS_OP_N, t.size, t.size,
                                        &one, t.data, t.size, t.vector, 1, &zero,
                                        t.result, 1));
        }
    }
    pthread_exit(NULL);
}
#else
template <typename T>
void execute(Task<T> &t, hipblasHandle_t *handle, hipStream_t *stream, int tid) {
    if (t.size < 100) {
        // perform on host
        printf("Task [%d], thread [%d] executing on host (%d)\n", t.id, tid, t.size);

        checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.data, 0, hipMemAttachHost));
        checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.vector, 0, hipMemAttachHost));
        checkCudaErrors(hipStreamAttachMemAsync(stream[0], t.result, 0, hipMemAttachHost));
        checkCudaErrors(hipStreamSynchronize(stream[0]));

        gemv(t.size, t.size, 1.0, t.data, t.vector, 0.0, t.result);
    }else {
        // perform on device
        printf("Task [%d], thread [%d] executing on device (%d)\n", t.id, tid, t.size);
        double one = 1.0;
        double zero = 0.0;

        checkCudaErrors(hipblasSetStream(handle[tid + 1], stream[tid + 1]));
        checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.data, 0, hipMemAttachSingle));
        checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.vector, 0, hipMemAttachSingle));
        checkCudaErrors(hipStreamAttachMemAsync(stream[tid + 1], t.result, 0, hipMemAttachSingle));
        // call the device operation
        checkCudaErrors(hipblasDgemv(handle[tid + 1], HIPBLAS_OP_N, t.size, t.size,
                                    &one, t.data, t.size, t.vector, 1, &zero,
                                    t.result, 1));
    }
}
#endif

// populate a list of tasks with random sizes
template <typename T>
void initialise_tasks(std::vector<Task<T> > &TaskList) {
    for (unsigned int i = 0; i < TaskList.size(); i++) {
        // generate random size
        int size;
        size = std::max((int)(drand48() * 1000.0), 64);
        TaskList[i].allocate(size, i);
    }
}

int main(int argc, char **argv) {
    // set device
    hipDeviceProp_t device_prop;
    int dev_id = findCudaDevice(argc, (const char **)argv);
    checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

    if (!device_prop.managedMemory) {
        // This samples requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");
        exit(EXIT_WAIVED);
    }

    if (device_prop.computeMode == hipComputeModeProhibited) {
        // This sample requires being run with a default or process exclusive mode
        fprintf(stderr,
                "This sample requires a device in either default or process "
                "exclusive mode\n");

        exit(EXIT_WAIVED);
    }

    // randomise task sizes
    int seed = (int)time(NULL);
    srand48(seed);

    // set number of threads
    const int nthreads = 4;

    // number of streams = number of threads
    hipStream_t *streams = new hipStream_t[nthreads + 1];
    hipblasHandle_t *handles = new hipblasHandle_t[nthreads + 1];

    for (int i = 0; i < nthreads + 1; ++i) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
        checkCudaErrors(hipblasCreate(&handles[i]));
    }

    // create list of N tasks
    unsigned int N = 40;
    std::vector<Task<double> > TaskList(N);
    initialise_tasks(TaskList);

    printf("Executing tasks on host / device. \n");

// run through all tasks using threads and streams
#ifdef USE_PTHREADS
    pthread_t threads[nthreads];
    threadData *InputToThreads = new threadData[nthreads];

    for (int i = 0; i < nthreads; ++i) {
        checkCudaErrors(hipSetDevice(dev_id));
        InputToThreads[i].tid = i;
        InputToThreads[i].streams = stream;
        InputToThreads[i].handles = handles;

        if ((TaskList.size() / nthreads) == 0) {
            InputToThreads[i].taskSize = (TaskList.size() / nthreads);
            InputToThreads[i].TaskListPtr = &TaskList[i * (TaskList.size() / nthreads)];
        }else {
            if (i == nthreads - 1) {
                InputToThreads[i].taskSize =
                    (TaskList.size() / nthreads) + (TaskList.size() % nthreads);
                InputToThreads[i].TaskListPtr =
                    &TaskList[i * (TaskList.size() / nthreads) +
                            (TaskList.size() % nthreads)];
            } else {
                InputToThreads[i].taskSize = (TaskList.size() / nthreads);
                InputToThreads[i].TaskListPtr =
                    &TaskList[i * (TaskList.size() / nthreads)];
            }
        }

        pthread_create(&threads[i], NULL, &execute, &InputToThreads[i]);
    }

    for (int i = 0; i < nthreads; ++i) {
        pthread_join(threads[i], NULL);
    }
#else
    // OpenMP 用于管理多线程并行,负责任务的动态分配
    omp_set_num_threads(nthreads);
#pragma omp parallel for schedule(dynamic)
    for (int i = 0; i < TaskList.size(); ++i) {
        checkCudaErrors(hipSetDevice(dev_id));
        int tid = omp_get_thread_num();
        execute(TaskList[i], handles, streams, tid);
    }
#endif

    hipDeviceSynchronize();

    // Destroy CUDA Streams, cuBlas handles
    for (int i = 0; i < nthreads + 1; i++) {
        hipStreamDestroy(streams[i]);
        hipblasDestroy(handles[i]);
    }

    // Free TaskList
    std::vector<Task<double> >().swap(TaskList);

    printf("All Done!\n");
    exit(EXIT_SUCCESS);
}
