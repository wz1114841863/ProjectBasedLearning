#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

namespace cg = cooperative_groups;

__global__ void clock_block(clock_t *d_o, clock_t clock_count) {
    unsigned int start_clock = (unsigned int)clock();
    clock_t clock_offset = 0;

    while (clock_offset < clock_count) {
        unsigned int end_clock = (unsigned int)clock();
        clock_offset = (clock_t)(end_clock - start_clock);
    }
    d_o[0] = clock_offset;
}

__global__ void sum(clock_t *d_clocks, int N) {
    cg::thread_block cta = cg::this_thread_block();
    __shared__ clock_t s_clocks[32];

    clock_t my_sum = 0;
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        my_sum += d_clocks[i];
    }

    s_clocks[threadIdx.x] = my_sum;
    cg::sync(cta);

    for (int i = 16; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
        }
        cg::sync(cta);
    }
    d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv) {
    int nkernels = 8;             // number of concurrent kernels
    int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
    int nbytes = nkernels * sizeof(clock_t);  // number of data bytes
    float kernel_time = 10;                   // time the kernel should run in ms
    float elapsed_time;                       // timing variables
    int cuda_device = 0;

    printf("[%s] - Starting...\n", argv[0]);

    // get number of kernels if overridden on the command line
    if (checkCmdLineFlag(argc, (const char **)argv, "nkernels")) {
        nkernels = getCmdLineArgumentInt(argc, (const char **)argv, "nkernels");
        nstreams = nkernels + 1;
    }

    cuda_device = findCudaDevice(argc, (const char **)argv);
    hipDeviceProp_t device_prop;
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&device_prop, cuda_device));
    if ((device_prop.concurrentKernels == 0)) {
        printf("> GPU does not support concurrent kernel execution\n");
        printf("  CUDA kernel runs will be serialized\n");
    }
    printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
        device_prop.major, device_prop.minor, device_prop.multiProcessorCount);

    clock_t *a = 0;
    checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
    clock_t *d_a = 0;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));

    hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    for (int i = 0; i < nstreams; ++i) {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));

    hipEvent_t *kernel_event;
    kernel_event = (hipEvent_t *)malloc(nkernels * sizeof(hipEvent_t));
    for (int i = 0; i < nkernels; ++i) {
        checkCudaErrors(hipEventCreateWithFlags(&(kernel_event[i]), hipEventDisableTiming));
    }

    // time execution with nkernels streams
    clock_t total_clocks = 0;
#if defined(__arm__) || defined(__aarch64__)
    // the kernel takes more time than the channel reset time on arm archs, so to
    // prevent hangs reduce time_clocks.
    clock_t time_clocks = (clock_t)(kernel_time * (device_prop.clockRate / 100));
#else
    clock_t time_clocks = (clock_t)(kernel_time * device_prop.clockRate);
#endif

    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done.
    for (int i = 0; i < nkernels; ++i) {
        clock_block<<<1, 1, 0, streams[i]>>>(&d_a[i], time_clocks);
        total_clocks += time_clocks;
        checkCudaErrors(hipEventRecord(kernel_event[i], streams[i]));
        checkCudaErrors(hipStreamWaitEvent(streams[nstreams - 1], kernel_event[i], 0));
    }

    sum<<<1, 32, 0, streams[nstreams - 1]>>>(d_a, nkernels);
    checkCudaErrors(hipMemcpyAsync(
        a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams - 1]));

    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

    printf("Expected time for serial execution of %d kernels = %.3fs\n", nkernels,
            nkernels * kernel_time / 1000.0f);
    printf("Expected time for concurrent execution of %d kernels = %.3fs\n",
            nkernels, kernel_time / 1000.0f);
    printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

    bool test_result = (a[0] > total_clocks);

    for (int i = 0; i < nkernels; i++) {
        hipStreamDestroy(streams[i]);
        hipEventDestroy(kernel_event[i]);
    }

    free(streams);
    free(kernel_event);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);

    if (!test_result) {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }

    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
